#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstring>

#include <nvtx3/nvToolsExt.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

class parser{
public:
    parser(int argc, char** argv){
        this->_grid_size = 512;
        this->_accur = 1e-6;
        this->_iters = 1000000;
        for (int i=0; i<argc-1; i++){
            std::string arg = argv[i];
            if (arg == "-accur"){
                std::string dump = std::string(argv[i+1]);
                this->_accur = std::stod(dump);
            }else if (arg == "-a"){
                this->_grid_size = std::stoi(argv[i + 1]);
            }else if (arg == "-i"){
                this->_iters = std::stoi(argv[i + 1]);
            }
        }

    };
    __host__ double accuracy() const{
        return this->_accur;
    }
    __host__ int iterations() const{
        return this->_iters;
    }
    __host__ int grid()const{
        return this->_grid_size;
    }
private:
    double _accur;
    int _grid_size;
    int _iters;

};

double corners[4] = {10, 20, 30, 20};

__global__
void cross_calc(double* A_kernel, double* B_kernel, size_t size){
    // get the block and thread indices
    
    size_t j = blockIdx.x;
    size_t i = threadIdx.x;
    // main cross computation. the average of 4 incident cells is taken
    if (i != 0 && j != 0){
       
        B_kernel[j * size + i] = 0.25 * (
            A_kernel[j * size + i - 1] + 
            A_kernel[j * size + i + 1] + 
            A_kernel[(j + 1) * size + i] + 
            A_kernel[(j - 1) * size + i]
        );
    
    }

}

__global__
void get_error_matrix(double* A_kernel, double* B_kernel, double* out){
    // get thread index
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    // take the difference between B_kernel and A_kernel
    if (blockIdx.x != 0 && threadIdx.x != 0){
        
        out[idx] = std::abs(B_kernel[idx] - A_kernel[idx]);
    
    }

}


int main(int argc, char ** argv){
    parser input = parser(argc, argv);

    int size = input.grid();
    double min_error = input.accuracy();
    int max_iter = input.iterations();
    int full_size = size * size;
    double step = (corners[1] - corners[0]) / (size - 1);
    // Matrixes initialization
    auto* A_kernel = new double[size * size];
    auto* B_kernel = new double[size * size];

    std::memset(A_kernel, 0, sizeof(double) * size * size);


    A_kernel[0] = corners[0];
    A_kernel[size - 1] = corners[1];
    A_kernel[size * size - 1] = corners[2];
    A_kernel[size * (size - 1)] = corners[3];



    for (int i = 1; i < size - 1; i ++) {
        A_kernel[i] = corners[0] + i * step;
        A_kernel[size * i] = corners[0] + i * step;
        A_kernel[(size-1) + size * i] = corners[1] + i * step;
        A_kernel[size * (size-1) + i] = corners[3] + i * step;
    }

    std::memcpy(B_kernel, A_kernel, sizeof(double) * full_size);
    // matrix output before computations
    // for (int i = 0; i < size; i ++) {
    //     for (int j = 0; j < size; j ++) {
    //         std::cout << A_kernel[j * size + i] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;
    
    // Choosing the device
    hipSetDevice(3);
    
    double* dev_A, *dev_B, *dev_err, *dev_err_mat, *temp_stor = NULL;
    size_t tmp_stor_size = 0;
    // Memory allocation for 2 matrixes and error variable on the device 
    hipError_t status_A = hipMalloc(&dev_A, sizeof(double) * full_size);
    hipError_t status_B = hipMalloc(&dev_B, sizeof(double) * full_size);
    hipError_t status = hipMalloc(&dev_err, sizeof(double));
    // some memory allocation accertions to catch errors
    if (status != hipSuccess){
        std::cout << "Device error variable allocation error " << status << std::endl;
        return status;
    }
    // memory allocation on device for error matrix
    status = hipMalloc(&dev_err_mat, sizeof(double) * full_size);
    if (status != hipSuccess){
        std::cout << "Device error matrix allocation error " << status << std::endl;
        return status;
    }
    if (status_A != hipSuccess){
        std::cout << "Kernel A allocation error " << status << std::endl;
        return status;
    } else if (status_B != hipSuccess){
        std::cout << "Kernel B allocation error " << status << std::endl;
        return status;
    }

    status_A = hipMemcpy(dev_A, A_kernel, sizeof(double) * full_size, hipMemcpyHostToDevice);
    if (status_A != hipSuccess){
        std::cout << "Kernel A copy to device error " << status << std::endl;
        return status_A;
    }
    status_B = hipMemcpy(dev_B, B_kernel, sizeof(double) * full_size, hipMemcpyHostToDevice);
    if (status_B != hipSuccess){
        std::cout << "kernel B copy to device error " << status << std::endl;
        return status_B;
    }

    status = hipcub::DeviceReduce::Max(temp_stor, tmp_stor_size, dev_err_mat, dev_err, full_size);
    if (status != hipSuccess){
        std::cout << "Max reduction error " << status << std::endl;
        return status;
    }

    status = hipMalloc(&temp_stor, tmp_stor_size);
    if (status != hipSuccess){
        std::cout << "Temporary storage allocation error " << status  << std::endl;
        return status;
    }

    int i = 0;
    double error = 1.0;
    // openining the nvtx mark for profiling
    nvtxRangePushA("Main loop");
    // main loop
    while (i < max_iter && error > min_error){
        i++;
        // compute one cross compurtation
        cross_calc<<<size-1, size-1>>>(dev_A, dev_B, size);

        if (i % 100 == 0){
            // get the error matrix. the difference between the matrixes
            // number of threads = (size-1)^2
            get_error_matrix<<<size - 1, size - 1>>>(dev_A, dev_B, dev_err_mat);
            // find the maximum error. result in dev_err
            hipcub::DeviceReduce::Max(temp_stor, tmp_stor_size, dev_err_mat, dev_err, full_size);
            // copying the error from device to host memory
            hipMemcpy(&error, dev_err, sizeof(double), hipMemcpyDeviceToHost);

        }
        // matrix swapping
        std::swap(dev_A, dev_B);


    }
    // closing the nvtx mark
    nvtxRangePop();
    // matrix output check after the computations
    // hipMemcpy(A_kernel, dev_A, sizeof(double) * full_size, hipMemcpyDeviceToHost);
    
    // for (int i = 0; i < size; i ++) {
    //     for (int j = 0; j < size; j ++) {
    //         std::cout << A_kernel[j * size + i] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // print out the results
    std::cout << "Error: " << error << std::endl;
    std::cout << "Iteration: " << i << std::endl;
    // memory deallocation
    hipFree(temp_stor);
    hipFree(dev_err_mat);
    hipFree(dev_A);
    hipFree(dev_B);
    delete[] A_kernel;
    delete[] B_kernel;
    return 0;
}
